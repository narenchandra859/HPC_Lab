
#include <hip/hip_runtime.h>
#include <stdio.h>
#define NUM_BLOCK 3	
#define BLOCK_WIDTH 5

__global__ void hello() {
	printf("\nHello world. Thread %d in block %d", threadIdx.x, blockIdx.x);
}

int main(int argc, char** argv) {
	hello<<<NUM_BLOCK, BLOCK_WIDTH>>>();
	hipDeviceSynchronize();
	printf("\nDone");
	return 0;
}