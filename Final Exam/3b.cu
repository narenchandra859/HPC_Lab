
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#define BLOCK_SIZE 3

int w = 6, h = 6;
int size = w*h;
int memSize = sizeof(float)*size;

__global__ void matrixMul(float *a, float *b, float *c, int w, int h) {
  int tx = (blockIdx.x*blockDim.x)+threadIdx.x;
  int ty = (blockIdx.y*blockDim.y)+threadIdx.y;
  float v = 0;
  int i;
  for(i = 0; i < w; i++)
    v += a[ty * w + i] * b[i * h + tx];
  c[ty * w + tx] = v;
}

void printMat(float *arr) {
  int i;
  for(i=0;i<size;i++) {
    printf("%f ", arr[i]);
    if((i+1)%w==0) printf("\n");
  }
}

int main(int argc, char** argv) {
  float *ha, *hb, *hc;
  int i;
  ha = (float*)malloc(memSize);
  hb = (float*)malloc(memSize);
  hc = (float*)malloc(memSize);
  for(i=0; i<size; i++)
    ha[i] = hb[i] = i;
  printf("\nMatrix A:\n");
  printMat(ha);
  printf("\nMatrix B:\n");
  printMat(hb);
  float *da, *db, *dc;
  hipMalloc(&da, memSize);
  hipMalloc(&db, memSize);
  hipMalloc(&dc, memSize);
  hipMemcpy(da, ha, memSize, hipMemcpyHostToDevice);
  hipMemcpy(db, hb, memSize, hipMemcpyHostToDevice);
  dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid(w/threads.x, h/threads.y);
  matrixMul<<<grid, threads>>>(da, db, dc, w, w);
  hipMemcpy(hc, dc, memSize, hipMemcpyDeviceToHost);
  printf("\nMatrix C:\n");
  printMat(hc);
  hipFree(da);hipFree(db);hipFree(dc);
  free(ha);free(hb);free(hc);
  return 0;
}
