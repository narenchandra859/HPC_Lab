
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void hello() {
	printf("\nHello from thread %d in block %d", threadIdx.x, blockIdx.x);
}
int main(int argc, char** argv) {
	hello<<<3,5>>>();
	hipDeviceSynchronize();
	printf("\n-------------------------------\n");
	hello<<<5,3>>>();
	hipDeviceSynchronize();
	return 0;
}