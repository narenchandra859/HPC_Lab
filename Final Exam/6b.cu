
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void vecAdd(double *a, double *b, double *c, int n) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	c[idx] = a[idx] + b[idx];
}

int main(int argc, char** argv) {
	double *ha, *hb, *hc;
	double *da, *db, *dc;
	int i, n = 100;
	size_t memsize = sizeof(double)*n;
	ha = (double *)malloc(memsize);
	hb = (double *)malloc(memsize);
	hc = (double *)malloc(memsize);
	for(i = 0; i < n; i++) {
		ha[i] = rand() % 10000;
		hb[i] = rand() % 10000;
	}
	hipMalloc(&da, memsize);
	hipMalloc(&db, memsize);
	hipMalloc(&dc, memsize);
	hipMemcpy(da, ha, memsize, hipMemcpyHostToDevice);
	hipMemcpy(db, hb, memsize, hipMemcpyHostToDevice);
	vecAdd<<<n/10, 10>>>(da, db, dc, n);
	hipMemcpy(hc, dc, memsize, hipMemcpyDeviceToHost);
	printf("\nResults: \n");
	for(i = 0; i < n; i++)
		printf("%f + %f = %f\n", ha[i], hb[i], hc[i]);
	hipFree(da);hipFree(db);hipFree(dc);
	free(ha);free(hb);free(hc);
	hipDeviceSynchronize();
	return 0;
}