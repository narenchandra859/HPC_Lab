
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void local_mem_GPU(float i) {
	float f;	// f is local
	f = i;		// f and i private to EACH thread
	printf("\nMy f value: %f", f);
}

__global__ void global_mem_GPU(float *arr) {
	// array is ptr to global memory on device
	arr[threadIdx.x] = 2.0f * (float) threadIdx.x;
}

__global__ void shared_mem_GPU(float *arr) {
	// local vars private to each thread
	int i, idx = threadIdx.x;
	float avg, sum = 0.0f;
	//shared visible to all threads in thread block
	__shared__ float sh_arr[128];
	//global arr to shared mem arr
	//each thread copies ONE element at idx
	sh_arr[idx] = arr[idx];
	__syncthreads(); //all writes completed check
	for(i=0; i<idx; i++)
		sum += sh_arr[i];
	avg = sum/(idx+1.0f);
	if(arr[idx]>avg)
		arr[idx] = avg;
	//above change can be seen by HOST/other thread blocks
}

int main(int argc, char** argv) {
	int i;
	// demonstrate local mem
	local_mem_GPU<<<1, 128>>>(2.0f);
	// demonstrate global mem
	float h_arr[128]; // host
	float *d_arr;
	hipMalloc(&d_arr, sizeof(float)*128);
	hipMemcpy(d_arr, h_arr, sizeof(float)*128, hipMemcpyHostToDevice);
	global_mem_GPU<<<1, 128>>>(d_arr);
	hipMemcpy(h_arr, d_arr, sizeof(float)*128, hipMemcpyDeviceToHost);	
	printf("\nAfter global execution\n");
	for(i=0; i<128; i++)
		printf(" %lf ", h_arr[i]);
	shared_mem_GPU<<<1, 128>>>(d_arr);
	hipMemcpy(h_arr, d_arr, sizeof(float)*128, hipMemcpyDeviceToHost);
	printf("\nAfter shared execution\n");
	for(i=0; i<128; i++)
		printf(" %lf ", h_arr[i]);
	hipDeviceSynchronize();
	return 0;
}
