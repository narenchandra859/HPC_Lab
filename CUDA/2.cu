#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void local_mem_GPU(float i) {
<<<<<<< Updated upstream
	float f;	// f is local
	f = i;		// f and i private to EACH thread
=======
	float f;
	f = i;		
>>>>>>> Stashed changes
	printf("\nMy f value: %f", f);
}

__global__ void global_mem_GPU(float *arr) {
<<<<<<< Updated upstream
	// array is ptr to global memory on device
=======
>>>>>>> Stashed changes
	arr[threadIdx.x] = 2.0f * (float) threadIdx.x;
}

__global__ void shared_mem_GPU(float *arr) {
<<<<<<< Updated upstream
	// local vars private to each thread
	int i, idx = threadIdx.x;
	float avg, sum = 0.0f;
	//shared visible to all threads in thread block
	__shared__ float sh_arr[128];
	//global arr to shared mem arr
	//each thread copies ONE element at idx
	sh_arr[idx] = arr[idx];
	__syncthreads(); //all writes completed check
=======
	int i, idx = threadIdx.x;
	float avg, sum = 0.0f;
	__shared__ float sh_arr[128];
	sh_arr[idx] = arr[idx];
	__syncthreads(); 
>>>>>>> Stashed changes
	for(i=0; i<idx; i++)
		sum += sh_arr[i];
	avg = sum/(idx+1.0f);
	if(arr[idx]>avg)
		arr[idx] = avg;
<<<<<<< Updated upstream
	//above change can be seen by HOST/other thread blocks
=======
>>>>>>> Stashed changes
}

int main(int argc, char** argv) {
	int i;
<<<<<<< Updated upstream
	// demonstrate local mem
	local_mem_GPU<<<1, 128>>>(2.0f);
	// demonstrate global mem
	float h_arr[128]; // host
=======
	local_mem_GPU<<<1, 128>>>(2.0f);
	float h_arr[128];
>>>>>>> Stashed changes
	float *d_arr;
	hipMalloc(&d_arr, sizeof(float)*128);
	hipMemcpy(d_arr, h_arr, sizeof(float)*128, hipMemcpyHostToDevice);
	global_mem_GPU<<<1, 128>>>(d_arr);
	hipMemcpy(h_arr, d_arr, sizeof(float)*128, hipMemcpyDeviceToHost);	
	printf("\nAfter global execution\n");
	for(i=0; i<128; i++)
		printf(" %lf ", h_arr[i]);
	shared_mem_GPU<<<1, 128>>>(d_arr);
	hipMemcpy(h_arr, d_arr, sizeof(float)*128, hipMemcpyDeviceToHost);
	printf("\nAfter shared execution\n");
	for(i=0; i<128; i++)
		printf(" %lf ", h_arr[i]);
	hipDeviceSynchronize();
	return 0;
<<<<<<< Updated upstream
}
=======
}
>>>>>>> Stashed changes
