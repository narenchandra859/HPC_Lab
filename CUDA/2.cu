
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void local_mem_GPU(float i) {
	float f;
	f = i;		
	printf("\nMy f value: %f", f);
}

__global__ void global_mem_GPU(float *arr) {
	arr[threadIdx.x] = 2.0f * (float) threadIdx.x;
}

__global__ void shared_mem_GPU(float *arr) {
	int i, idx = threadIdx.x;
	float avg, sum = 0.0f;
	__shared__ float sh_arr[128];
	sh_arr[idx] = arr[idx];
	__syncthreads(); 
	for(i=0; i<idx; i++)
		sum += sh_arr[i];
	avg = sum/(idx+1.0f);
	if(arr[idx]>avg)
		arr[idx] = avg;
}

int main(int argc, char** argv) {
	int i;
	local_mem_GPU<<<1, 128>>>(2.0f);
	float h_arr[128];
	float *d_arr;
	hipMalloc(&d_arr, sizeof(float)*128);
	hipMemcpy(d_arr, h_arr, sizeof(float)*128, hipMemcpyHostToDevice);
	global_mem_GPU<<<1, 128>>>(d_arr);
	hipMemcpy(h_arr, d_arr, sizeof(float)*128, hipMemcpyDeviceToHost);	
	printf("\nAfter global execution\n");
	for(i=0; i<128; i++)
		printf(" %lf ", h_arr[i]);
	shared_mem_GPU<<<1, 128>>>(d_arr);
	hipMemcpy(h_arr, d_arr, sizeof(float)*128, hipMemcpyDeviceToHost);
	printf("\nAfter shared execution\n");
	for(i=0; i<128; i++)
		printf(" %lf ", h_arr[i]);
	hipDeviceSynchronize();
	return 0;
}