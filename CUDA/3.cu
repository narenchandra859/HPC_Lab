
#include <hip/hip_runtime.h>
#include <stdio.h>
#define NUM_BLOCKS 32
#define BLOCK_WIDTH 5
__global__ void hello() {
	printf("\nThread %d in block %d", threadIdx.x, blockIdx.x);
}
int main(int argc, char** argv) {
	hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();
	hipDeviceSynchronize();
	printf("\nDone\n");
	return 0;
}
