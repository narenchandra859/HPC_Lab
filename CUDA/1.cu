#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

<<<<<<< Updated upstream
__global__ void vecAdd(double *a, double *b, double *c, int n) {
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	if (id < n) c[id] = a[id] + b[id];
}

int main(int argc, char** argv) {
	int n = 100;
	double *ha, *hb, *hc, *da, *db, *dc;
	size_t bytes = n*sizeof(double);
	ha = (double*)malloc(bytes);
	hb = (double*)malloc(bytes);
	hc = (double*)malloc(bytes);
	hipMalloc(&da, bytes);
	hipMalloc(&db, bytes);
	hipMalloc(&dc, bytes);
	int i;
	for(i = 0; i < n; i++)
		ha[i] = hb[i] = i;
	hipMemcpy(da, ha, bytes, hipMemcpyHostToDevice);
	hipMemcpy(db, hb, bytes, hipMemcpyHostToDevice);
	int blockSize, gridSize;
	blockSize = 1024;
	gridSize = (int)ceil((float)n/blockSize);
	vecAdd<<<gridSize, blockSize>>>(da, db, dc, n);
	hipMemcpy(hc, dc, bytes, hipMemcpyDeviceToHost);
	for(i = 0; i < n; i++)
		printf("\n%f + %f = %f\n",ha[i], hb[i], hc[i]);
	hipFree(da);
	hipFree(db);
	hipFree(dc);
	free(ha);
	free(hb);
	free(hc);
	return 0;
}
=======
__global__ void vecAdd(double * a, double * b, double * c, int n) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
        c[id] = a[id] + b[id];
}

int main(int argc, char * argv[]) {
    int n = 100, i;
    double *h_a, *h_b, *h_c;
    double *d_a, *d_b, *d_c;
    size_t bytes = n * sizeof(double);
    h_a = (double*)malloc(bytes);
    h_b = (double*)malloc(bytes);
    h_c = (double*)malloc(bytes);
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
    for (i = 0; i < n; i++) 
        h_a[i] = h_b[i] = i;
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
    int blockSize, gridSize;
    blockSize = 1024;
    gridSize = (int) ceil((float) n / blockSize);
    vecAdd <<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
    for (i = 0; i < n; i++)
        printf(" %f + %f =%f\n", h_a[i], h_b[i], h_c[i]);
    hipFree(d_a);hipFree(d_b);hipFree(d_c);
    free(h_a);free(h_b);free(h_c);
    return 0;
}
>>>>>>> Stashed changes
