
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void vecAdd(double *a, double *b, double *c, int n) {
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	if (id < n) c[id] = a[id] + b[id];
}

int main(int argc, char** argv) {
	int n = 100;
	double *ha, *hb, *hc, *da, *db, *dc;
	size_t bytes = n*sizeof(double);
	ha = (double*)malloc(bytes);
	hb = (double*)malloc(bytes);
	hc = (double*)malloc(bytes);
	hipMalloc(&da, bytes);
	hipMalloc(&db, bytes);
	hipMalloc(&dc, bytes);
	int i;
	for(i = 0; i < n; i++)
		ha[i] = hb[i] = i;
	hipMemcpy(da, ha, bytes, hipMemcpyHostToDevice);
	hipMemcpy(db, hb, bytes, hipMemcpyHostToDevice);
	int blockSize, gridSize;
	blockSize = 1024;
	gridSize = (int)ceil((float)n/blockSize);
	vecAdd<<<gridSize, blockSize>>>(da, db, dc, n);
	hipMemcpy(hc, dc, bytes, hipMemcpyDeviceToHost);
	for(i = 0; i < n; i++)
		printf("\n%f + %f = %f\n",ha[i], hb[i], hc[i]);
	hipFree(da);
	hipFree(db);
	hipFree(dc);
	free(ha);
	free(hb);
	free(hc);
	return 0;
}
